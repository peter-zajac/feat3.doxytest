// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2020 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/transpose.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

void Transpose<Mem::CUDA>::value(float * r, const float * const x, Index rows_x, Index columns_x)
{
  hipblasStatus_t status;
  float one(1);
  float zero(0);

  if (r == x)
  {
    float * temp;
    hipMalloc((void**)&temp, rows_x * columns_x * sizeof(float));
    hipMemcpy(temp, x, rows_x * columns_x * sizeof(float), hipMemcpyDeviceToDevice);
    status = hipblasSgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows_x, columns_x, &one, temp, columns_x, &zero, nullptr, columns_x, r, rows_x);
  }
  else
  {
    status = hipblasSgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows_x, columns_x, &one, x, columns_x, &zero, nullptr, columns_x, r, rows_x);
  }
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "hipblasSgeam failed with status code: " + stringify(status));

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

void Transpose<Mem::CUDA>::value(double * r, const double * const x, Index rows_x, Index columns_x)
{
  hipblasStatus_t status;
  double one(1);
  double zero(0);

  if (r == x)
  {
    double * temp;
    hipMalloc((void**)&temp, rows_x * columns_x * sizeof(double));
    hipMemcpy(temp, x, rows_x * columns_x * sizeof(double), hipMemcpyDefault);
    status = hipblasDgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows_x, columns_x, &one, temp, columns_x, &zero, nullptr, columns_x, r, rows_x);
  }
  else
  {
    status = hipblasDgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows_x, columns_x, &one, x, columns_x, &zero, nullptr, columns_x, r, rows_x);
  }

  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "hipblasDgeam failed with status code: " + stringify(status));

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
