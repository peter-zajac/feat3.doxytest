#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/norm.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      float cuda_norm2(const float * x, const Index size)
      {
        float result;
        hipblasStatus_t status;
        status = hipblasSnrm2(Util::Intern::cublas_handle, size, x, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cublasnrm2 failed with status code: " + stringify(status));
        return result;
      }

      double cuda_norm2(const double * x, const Index size)
      {
        double result;
        hipblasStatus_t status;
        status = hipblasDnrm2(Util::Intern::cublas_handle, size, x, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cublasnrm2 failed with status code: " + stringify(status));
        return result;
      }
    }
  }
}

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
DT_ Norm2<Mem::CUDA>::value(const DT_ * const x, const Index size)
{
  DT_ result = Intern::cuda_norm2(x, size);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  return result;
}

template float Norm2<Mem::CUDA>::value(const float * const, const Index);
template double Norm2<Mem::CUDA>::value(const double * const, const Index);
