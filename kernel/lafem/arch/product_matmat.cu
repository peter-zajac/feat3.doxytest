// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/product_matmat.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipblas.h"

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      void cublas_product_matmat_dense(int m, int n, int k,
                                       const float * x,
                                       const float * y, float * r)
      {
        hipblasStatus_t status;
        const float one(1.f);
        const float zero(0.f);
        float * temp;
        hipMalloc((void**)&temp, m*n*sizeof(float));

        status = hipblasSgemm(Util::Intern::cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &one, y, n, x, k, &zero, r, n);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasSgemm failed with status code: " + stringify(status));

        hipFree(temp);
      }

      void cublas_product_matmat_dense(int m, int n, int k,
                                       const double * x,
                                       const double * y, double * r)
      {
        hipblasStatus_t status;
        const double one(1.);
        const double zero(0.);
        double * temp;
        hipMalloc((void**)&temp, m*n*sizeof(double));

        status = hipblasDgemm(Util::Intern::cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &one, y, n, x, k, &zero, r, n);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasDgemm failed with status code: " + stringify(status));

        hipFree(temp);
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void ProductMatMat<Mem::CUDA>::dense(DT_ * r, const DT_ * const x, const DT_ * const y, const Index rows, const Index columns, const Index inner)
{
  FEAT::LAFEM::Intern::cublas_product_matmat_dense((int)rows, (int)columns, (int)inner, x, y, r);

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void ProductMatMat<Mem::CUDA>::dense(float *, const float * const, const float * const, const Index, const Index, const Index);
template void ProductMatMat<Mem::CUDA>::dense(double *, const double * const, const double * const, const Index, const Index, const Index);
