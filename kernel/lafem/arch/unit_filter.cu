#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/unit_filter.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

/// \cond internal
namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_>
      __global__ void cuda_unit_filter_rhs(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;
        v[sv_indices[idx]] = sv_elements[idx];
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_unit_filter_def(DT_ * v, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;
        v[sv_indices[idx]] = DT_(0);
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_, typename IT_>
void UnitFilter::filter_rhs_cuda(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_rhs<<<grid, block>>>(v, sv_elements, sv_indices, ue);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilter::filter_rhs_cuda(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilter::filter_rhs_cuda(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilter::filter_rhs_cuda(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilter::filter_rhs_cuda(double *, const double * const, const unsigned int * const, const Index);

template <typename DT_, typename IT_>
void UnitFilter::filter_def_cuda(DT_ * v, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_def<<<grid, block>>>(v, sv_indices, ue);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilter::filter_def_cuda(float *, const unsigned long * const, const Index);
template void UnitFilter::filter_def_cuda(double *, const unsigned long * const, const Index);
template void UnitFilter::filter_def_cuda(float *, const unsigned int * const, const Index);
template void UnitFilter::filter_def_cuda(double *, const unsigned int * const, const Index);

/// \endcond
