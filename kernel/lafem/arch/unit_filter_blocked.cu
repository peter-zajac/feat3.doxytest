#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/unit_filter_blocked.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

/// \cond internal
namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_rhs(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        for(Index j(0) ; j < block_size; ++j)
          v[block_size* sv_indices[idx] + j] = sv_elements[block_size * idx + j];
      }

      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_def(DT_ * v, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        for(Index j(0) ; j < block_size; ++j)
          v[block_size * sv_indices[idx] + j] = DT_(0);
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_, typename IT_, int BlockSize_>
void UnitFilterBlocked<Mem::CUDA>::filter_rhs(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_blocked_rhs<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_elements, sv_indices, ue);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 1>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 1>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 1>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 1>(double *, const double * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 2>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 2>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 2>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 2>(double *, const double * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 3>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 3>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 3>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 3>(double *, const double * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned long, 4>(float *, const float * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned long, 4>(double *, const double * const, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<float, unsigned int, 4>(float *, const float * const, const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_rhs<double, unsigned int, 4>(double *, const double * const, const unsigned int * const, const Index);

template <typename DT_, typename IT_, int BlockSize_>
void UnitFilterBlocked<Mem::CUDA>::filter_def(DT_ * v, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_blocked_def<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_indices, ue);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 1>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 1>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 1>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 1>(double *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 2>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 2>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 2>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 2>(double *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 3>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 3>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 3>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 3>(double *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned long, 4>(float *, const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned long, 4>(double *,  const unsigned long * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<float, unsigned int, 4>(float *,  const unsigned int * const, const Index);
template void UnitFilterBlocked<Mem::CUDA>::filter_def<double, unsigned int, 4>(double *,  const unsigned int * const, const Index);

/// \endcond
