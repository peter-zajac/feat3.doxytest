#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2023 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/unit_filter_blocked.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

/// \cond internal
namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_rhs(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        for(Index j(0) ; j < block_size; ++j)
          v[block_size* sv_indices[idx] + j] = sv_elements[block_size * idx + j];
      }

      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_def(DT_ * v, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        for(Index j(0) ; j < block_size; ++j)
          v[block_size * sv_indices[idx] + j] = DT_(0);
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <int BlockSize_, typename DT_, typename IT_>
void UnitFilterBlocked::filter_rhs_cuda(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_blocked_rhs<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_elements, sv_indices, ue);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked::filter_rhs_cuda<1, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<1, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<1, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<1, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<2, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<2, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<2, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<2, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<3, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<3, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<3, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<3, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<4, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<4, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<4, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_rhs_cuda<4, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index);

template <int BlockSize_, typename DT_, typename IT_>
void UnitFilterBlocked::filter_def_cuda(DT_ * v, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_blocked_def<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_indices, ue);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked::filter_def_cuda<1, float, std::uint64_t>(float *, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<1, double, std::uint64_t>(double *,  const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<1, float, std::uint32_t>(float *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<1, double, std::uint32_t>(double *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<2, float, std::uint64_t>(float *, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<2, double, std::uint64_t>(double *,  const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<2, float, std::uint32_t>(float *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<2, double, std::uint32_t>(double *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<3, float, std::uint64_t>(float *, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<3, double, std::uint64_t>(double *,  const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<3, float, std::uint32_t>(float *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<3, double, std::uint32_t>(double *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<4, float, std::uint64_t>(float *, const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<4, double, std::uint64_t>(double *,  const std::uint64_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<4, float, std::uint32_t>(float *,  const std::uint32_t * const, const Index);
template void UnitFilterBlocked::filter_def_cuda<4, double, std::uint32_t>(double *,  const std::uint32_t * const, const Index);

/// \endcond
