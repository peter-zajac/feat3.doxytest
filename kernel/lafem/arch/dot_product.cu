// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/dot_product.hpp>
#include <kernel/lafem/arch/component_product.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {

      float cuda_dot_product(const float * x, const float * y, const Index size)
      {
        float result;
        hipblasStatus_t status;
        status = hipblasSdot(Util::Intern::cublas_handle, size, x, 1, y, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cublasdot failed with status code "+ stringify(status));
        return result;
      }

      double cuda_dot_product(const double * x, const double * y, const Index size)
      {
        double result;
        hipblasStatus_t status;
        status = hipblasDdot(Util::Intern::cublas_handle, size, x, 1, y, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cublasdot failed with status code "+ stringify(status));
        return result;
      }
    }
  }
}

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
DT_ DotProduct<Mem::CUDA>::value(const DT_ * const x, const DT_ * const y, const Index size)
{
  DT_ result = Intern::cuda_dot_product(x, y, size);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  return result;
}

template float DotProduct<Mem::CUDA>::value(const float * const, const float * const, const Index);
template double DotProduct<Mem::CUDA>::value(const double * const, const double * const, const Index);

template <typename DT_>
DT_ TripleDotProduct<Mem::CUDA>::value(const DT_ * const x, const DT_ * const y, const DT_ * const z, const Index size)
{
  DT_ * temp;
  hipMalloc((void **) &temp, size * sizeof(DT_));
  ComponentProduct<Mem::CUDA>::value(temp, y, z, size);
  DT_ result = Intern::cuda_dot_product(x, temp, size);
  hipFree(temp);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  return result;
}

template float TripleDotProduct<Mem::CUDA>::value(const float * const x, const float * const y, const float * const z, const Index size);
template double TripleDotProduct<Mem::CUDA>::value(const double * const x, const double * const y, const double * const z, const Index size);
