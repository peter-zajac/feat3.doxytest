#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/diagonal.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename IT_>
      __global__ void cuda_diagonal_csr(IT_ * diag, const IT_ * col_ind, const IT_ * row_ptr, const Index rows)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]); i < end; ++i)
        {
          if (idx == col_ind[i])
          {
            diag[idx] = i;
            return;
          }
        }
        diag[idx] = row_ptr[rows];
      }

      template <typename IT_>
      __global__ void cuda_diagonal_csrb(IT_ * diag, const IT_ * col_ind, const IT_ * row_ptr, const Index rows)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]); i < end; ++i)
        {
          if (idx == col_ind[i])
          {
            diag[idx] = i;
            return;
          }
        }
        diag[idx] = row_ptr[rows];
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_diagonal_ell(DT_ * diag, const DT_ * val, const IT_ * col_ind,
                                              const IT_ * cs, const IT_ * cl, const Index C, const Index rows)
      {
        const Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;


        const Index chunk(idx / C);
        const Index local_row(idx % C);
        const Index chunk_end(cs[chunk+1]);

        for (Index pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol+=C)
        {
          if (col_ind[pcol] == idx)
          {
            diag[idx] = val[pcol];
            return;
          }
        }
        diag[idx] = DT_(0);

      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename IT_>
void Diagonal<Mem::CUDA>::csr(IT_ * diag, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_diagonal_csr<<<grid, block>>>(diag, col_ind, row_ptr, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void Diagonal<Mem::CUDA>::csr(unsigned long *, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csr(unsigned int *,const unsigned int * const, const unsigned int * const, const Index);

template <typename IT_>
void Diagonal<Mem::CUDA>::csrb(IT_ * diag, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_diagonal_csrb<IT_><<<grid, block>>>(diag, col_ind, row_ptr, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Diagonal<Mem::CUDA>::csrb(unsigned long *, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb(unsigned int *, const unsigned int * const, const unsigned int * const, const Index);

template <typename DT_, typename IT_>
void Diagonal<Mem::CUDA>::ell(DT_ * diag, const DT_ * const val, const IT_ * const col_ind,
    const IT_ * const cs, const IT_ * const cl, const Index C, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_diagonal_ell<<<grid, block>>>(diag, val, col_ind, cs, cl, C, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void Diagonal<Mem::CUDA>::ell(float *, const float * const, const unsigned long * const,
    const unsigned long * const, const unsigned long * const, const Index, const Index);
template void Diagonal<Mem::CUDA>::ell(double *, const double * const, const unsigned long * const,
    const unsigned long * const, const unsigned long * const, const Index, const Index);
template void Diagonal<Mem::CUDA>::ell(float *, const float * const, const unsigned int * const,
    const unsigned int * const, const unsigned int * const, const Index, const Index);
template void Diagonal<Mem::CUDA>::ell(double *, const double * const, const unsigned int * const,
    const unsigned int * const, const unsigned int * const, const Index, const Index);
