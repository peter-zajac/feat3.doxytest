#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/diagonal.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_>
      __global__ void cuda_diagonal_csr(DT_ * diag, const DT_ * val, const IT_ * col_ind, const IT_ * row_ptr, const Index rows)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]); i < end; ++i)
        {
          if (idx == col_ind[i])
          {
            diag[idx] = val[i];
            return;
          }
        }
        diag[idx] = 0;
      }

      template <typename DT_, typename IT_, int BlockHeight_, int BlockWidth_>
      __global__ void cuda_diagonal_csrb(DT_ * diag, const DT_ * val, const IT_ * col_ind, const IT_ * row_ptr, const Index rows)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index row(idx);

        for (Index i(0) ; i < BlockHeight_ ; ++i)
        {
          diag[row * BlockHeight_ + i] = DT_(0);
        }
        const Index end = row_ptr[row + 1];

        for (Index col = row_ptr[row]; col < end; col++)
        {
          if (row == col_ind[col])
          {
            for (Index i(0) ; i < BlockHeight_ ; ++i)
            {
              diag[row * BlockHeight_ + i] = val[(col * BlockHeight_ * BlockWidth_) + i + i * BlockWidth_];
            }
          }
        }
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_diagonal_ell(DT_ * diag, const DT_ * val, const IT_ * col_ind,
                                              const IT_ * cs, const IT_ * cl, const Index C, const Index rows)
      {
        const Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;


        const Index chunk(idx / C);
        const Index local_row(idx % C);
        const Index chunk_end(cs[chunk+1]);

        for (Index pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol+=C)
        {
          if (col_ind[pcol] == idx)
          {
            diag[idx] = val[pcol];
            return;
          }
        }
        diag[idx] = DT_(0);

      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_, typename IT_>
void Diagonal<Mem::CUDA>::csr(DT_ * diag, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_diagonal_csr<<<grid, block>>>(diag, val, col_ind, row_ptr, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void Diagonal<Mem::CUDA>::csr(float *, const float * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csr(double *, const double * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csr(float *, const float * const, const unsigned int * const, const unsigned int * const, const Index);
template void Diagonal<Mem::CUDA>::csr(double *, const double * const, const unsigned int * const, const unsigned int * const, const Index);

template <typename DT_, typename IT_, int BlockHeight_, int BlockWidth_>
void Diagonal<Mem::CUDA>::csrb(DT_ * diag, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_diagonal_csrb<DT_, IT_, BlockHeight_, BlockWidth_><<<grid, block>>>(diag, val, col_ind, row_ptr, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Diagonal<Mem::CUDA>::csrb<float, unsigned long, 1, 1>(float *, const float * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<double, unsigned long, 1, 1>(double *, const double * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<float, unsigned int, 1, 1>(float *, const float * const, const unsigned int * const, const unsigned int * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<double, unsigned int, 1, 1>(double *, const double * const, const unsigned int * const, const unsigned int * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<float, unsigned long, 2, 2>(float *, const float * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<double, unsigned long, 2, 2>(double *, const double * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<float, unsigned int, 2, 2>(float *, const float * const, const unsigned int * const, const unsigned int * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<double, unsigned int, 2, 2>(double *, const double * const, const unsigned int * const, const unsigned int * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<float, unsigned long, 3, 3>(float *, const float * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<double, unsigned long, 3, 3>(double *, const double * const, const unsigned long * const, const unsigned long * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<float, unsigned int, 3, 3>(float *, const float * const, const unsigned int * const, const unsigned int * const, const Index);
template void Diagonal<Mem::CUDA>::csrb<double, unsigned int, 3, 3>(double *, const double * const, const unsigned int * const, const unsigned int * const, const Index);

template <typename DT_, typename IT_>
void Diagonal<Mem::CUDA>::ell(DT_ * diag, const DT_ * const val, const IT_ * const col_ind,
    const IT_ * const cs, const IT_ * const cl, const Index C, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_diagonal_ell<<<grid, block>>>(diag, val, col_ind, cs, cl, C, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void Diagonal<Mem::CUDA>::ell(float *, const float * const, const unsigned long * const,
    const unsigned long * const, const unsigned long * const, const Index, const Index);
template void Diagonal<Mem::CUDA>::ell(double *, const double * const, const unsigned long * const,
    const unsigned long * const, const unsigned long * const, const Index, const Index);
template void Diagonal<Mem::CUDA>::ell(float *, const float * const, const unsigned int * const,
    const unsigned int * const, const unsigned int * const, const Index, const Index);
template void Diagonal<Mem::CUDA>::ell(double *, const double * const, const unsigned int * const,
    const unsigned int * const, const unsigned int * const, const Index, const Index);
