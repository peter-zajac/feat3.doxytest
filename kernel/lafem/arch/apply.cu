#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/archs.hpp>
#include <kernel/lafem/arch/apply.hpp>
#include <kernel/lafem/arch/component_product.hpp>
#include <kernel/lafem/arch/scale.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>
#include <kernel/util/math.hpp>

#include "hipsparse.h"

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_apply_mv_csr(DT_ * r, const DT_ a, const DT_ * x, const DT_ b, const DT_ * val,
          const unsigned long * col_ind, const unsigned long * row_ptr, const Index count, const bool transposed)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;

        DT_ sum(0);
        const Index end(row_ptr[idx + 1]);
        for (Index i(row_ptr[idx]) ; i < end ; ++i)
        {
          sum += val[i] * x[col_ind[i]];
        }
        r[idx] = (sum * a) + b * r[idx];
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_apply_mv_ell(DT_ * r, const DT_ a, const DT_ * x, const DT_ b, const DT_ * val, const IT_ * col_ind,
                                       const IT_ * cs, const IT_ * cl, const Index rows, const Index C)
      {
        const Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;


        DT_ sum(0);
        const Index chunk(idx / C);
        const Index local_row(idx % C);
        const Index chunk_end(cs[chunk+1]);

        for (Index pcol(cs[chunk] + local_row) ; pcol < chunk_end ; pcol+=C)
        {
          sum += val[pcol] * x[col_ind[pcol]];
        }
        r[idx] = sum * a + b * r[idx];

      }

      template <typename DT_, typename IT_>
      __global__ void cuda_apply_banded(DT_ * r, const DT_ alpha, const DT_ * x, const DT_ beta, const DT_ * val, const IT_ * offsets, const Index num_of_offsets, const Index rows, const Index columns)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= rows)
          return;

        const Index k1(rows - 1);
        const Index k2(rows + columns - 1);

        Index start(0);

        while (k1 > offsets[start] + idx)
        {
          ++start;
        }

        Index end(start);

        while (end < num_of_offsets && idx + offsets[end] < k2)
        {
          ++end;
        }

        DT_ sum(DT_(0.0));
        for (Index diag(start); diag < end; ++diag)
        {
          sum += val[rows * diag + idx] * x[idx + offsets[diag] - rows + 1];
        }
        r[idx] = (sum*alpha) + beta * r[idx];
      }

      void cusparse_apply_csr(hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const float * alpha, const hipsparseMatDescr_t descrA,
                                       const float * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       const float * x, const float * beta, float * y)
      {
        hipsparseStatus_t status;
        status = hipsparseScsrmv(Util::Intern::cusparse_handle, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrmv failed with status code: " + stringify(status));
      }

      void cusparse_apply_csr(hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const double * alpha, const hipsparseMatDescr_t descrA,
                                       const double * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       const double * x, const double * beta, double * y)
      {
        hipsparseStatus_t status;
        status = hipsparseDcsrmv(Util::Intern::cusparse_handle, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrmv failed with status code: " + stringify(status));
      }

      void cusparse_apply_csrb(hipsparseDirection_t dir, hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const float * alpha, const hipsparseMatDescr_t descrA,
                                       const float * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       int block_dim,
                                       const float * x, const float * beta, float * y)
      {
        hipsparseStatus_t status;
        status = hipsparseSbsrmv(Util::Intern::cusparse_handle, dir, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, block_dim, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrmv failed with status code: " + stringify(status));
      }

      void cusparse_apply_csrb(hipsparseDirection_t dir, hipsparseOperation_t trans,
                                       int m, int n, int nnz,
                                       const double * alpha, const hipsparseMatDescr_t descrA,
                                       const double * csrVal, const int * csrRowPtr, const int *csrColInd,
                                       int block_dim,
                                       const double * x, const double * beta, double * y)
      {
        hipsparseStatus_t status;
        status = hipsparseDbsrmv(Util::Intern::cusparse_handle, dir, trans, m, n, nnz, alpha, descrA, csrVal, csrRowPtr,
                       csrColInd, block_dim, x, beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrmv failed with status code: " + stringify(status));
      }

      void cublas_apply_dense(hipblasOperation_t trans,
                                       int m, int n,
                                       const float * alpha,
                                       const float * val,
                                       const float * x, const float * beta, float * y)
      {
        hipblasStatus_t status;
        status = hipblasSgemv(Util::Intern::cublas_handle, trans, n, m, alpha, val, n, x, 1, beta, y, 1);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasSgemv failed with status code: " + stringify(status));
      }

      void cublas_apply_dense(hipblasOperation_t trans,
                                       int m, int n,
                                       const double * alpha,
                                       const double * val,
                                       const double * x, const double * beta, double * y)
      {
        hipblasStatus_t status;
        status = hipblasDgemv(Util::Intern::cublas_handle, trans, n, m, alpha, val, n, x, 1, beta, y, 1);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipblasDgemv failed with status code: " + stringify(status));
      }

      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_apply_csrsb(DT_ * r, const DT_ a, const DT_ * x, const DT_ b, const DT_ * val, const IT_ * col_ind,
                                              const IT_ * row_ptr, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;

        DT_ bsum[BlockSize_];
        for (int j(0) ; j < BlockSize_ ; ++j)
        {
          bsum[j] = DT_(0);
        }
        const IT_ end(row_ptr[idx + 1]);
        for (IT_ i(row_ptr[idx]) ; i < end ; ++i)
        {
          const DT_ vali(val[i]);
          const IT_ coli(col_ind[i] * BlockSize_);
          for (int j(0) ; j < BlockSize_ ; ++j)
          {
            bsum[j] += vali * x[coli + j];
          }
        }
        for (int j(0) ; j < BlockSize_ ; ++j)
        {
          r[idx * BlockSize_ + j] = (bsum[j] * a) + b * r[idx * BlockSize_ + j];
        }
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_apply_csrb(DT_ * r, const DT_ a, const DT_ * x, const DT_ b, const DT_ * val, const IT_ * col_ind,
          const IT_ * row_ptr, const Index count, const int BlockHeight, const int BlockWidth)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;

        /// \todo remove hardcoded number
        DT_ bsum[10];
        for (int j(0) ; j < BlockHeight ; ++j)
        {
          bsum[j] = DT_(0);
        }
        const unsigned long end(row_ptr[idx + 1]);
        for (unsigned long i(row_ptr[idx]) ; i < end ; ++i)
        {
          for (int h(0) ; h < BlockHeight ; ++h)
          {
            for (int w(0) ; w < BlockWidth ; ++w)
            {
              bsum[h] += val[i * BlockHeight * BlockWidth + h * BlockWidth + w] * x[col_ind[i] * BlockWidth + w];
            }
          }
        }
        for (int j(0) ; j < BlockHeight ; ++j)
        {
          r[idx * BlockHeight + j] = (bsum[j] * a) + b * r[idx * BlockHeight + j];
        }
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void Apply<Mem::CUDA>::csr(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const unsigned long * const col_ind, const unsigned long * const row_ptr, const Index rows, const Index columns, const Index used_elements, const bool transposed)
{
  if (transposed)
    throw InternalError(__func__, __FILE__, __LINE__, "transposedd csr product not supported for IT=unsigned long!");

  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  if (Math::abs(b) < Math::eps<DT_>())
  {
    MemoryPool<Mem::CUDA>::set_memory(r, DT_(0), (transposed?columns:rows));
  }
  else if (r != y)
  {
    MemoryPool<Mem::CUDA>::copy(r, y, (transposed?columns:rows));
  }

  FEAT::LAFEM::Intern::cuda_apply_mv_csr<<<grid, block>>>(r, a, x, b, val, col_ind, row_ptr, rows, transposed);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Apply<Mem::CUDA>::csr(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index, const bool);
template void Apply<Mem::CUDA>::csr(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index, const bool);

template <typename DT_>
void Apply<Mem::CUDA>::csr(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const unsigned int * const col_ind, const unsigned int * const row_ptr, const Index rows, const Index columns, const Index used_elements, const bool transposed)
{
  hipsparseOperation_t trans;
  if (transposed)
    trans = HIPSPARSE_OPERATION_TRANSPOSE;
  else
    trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  hipsparseMatDescr_t descr=0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  if (r != y)
  {
    MemoryPool<Mem::CUDA>::copy(r, y, (transposed?columns:rows));
  }

  FEAT::LAFEM::Intern::cusparse_apply_csr(trans, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind, x, &b, r);

  hipsparseDestroyMatDescr(descr);

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Apply<Mem::CUDA>::csr(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index, const bool);
template void Apply<Mem::CUDA>::csr(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index, const bool);

//silence the compiler by pretending to accept any IT_ but hopefully only 'unsigned int' calls will be made
//this circumnavigates the missing static_if in csrb_wrapper
template <typename DT_, typename IT_>
void Apply<Mem::CUDA>::csrb_intern(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows, const Index columns, const Index used_elements, const int BlockSize)
{
  if (r != y)
  {
    hipMemcpy(r, y, rows * BlockSize * sizeof(DT_), hipMemcpyDeviceToDevice);
  }

  hipsparseMatDescr_t descr=0;
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  FEAT::LAFEM::Intern::cusparse_apply_csrb(HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE, (int)rows, (int)columns, (int)used_elements, &a, descr, val, (int*)row_ptr, (int*)col_ind,
      BlockSize, x, &b, r);

  hipsparseDestroyMatDescr(descr);

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template <typename DT_, typename IT_>
void Apply<Mem::CUDA>::csrb_intern(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows, const Index columns, const Index used_elements, const int BlockHeight, const int BlockWidth)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  if (Math::abs(b) < Math::eps<DT_>())
  {
    MemoryPool<Mem::CUDA>::set_memory(r, DT_(0), /*(transposed?columns:rows)*/ rows * BlockHeight);
  }
  else if (r != y)
  {
    MemoryPool<Mem::CUDA>::copy(r, y, /*(transposed?columns:rows)*/ rows * BlockHeight);
  }

  FEAT::LAFEM::Intern::cuda_apply_csrb<DT_, IT_><<<grid, block>>>(r, a, x, b, val, col_ind, row_ptr, rows, BlockHeight, BlockWidth);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template <typename DT_, typename IT_>
void Apply<Mem::CUDA>:: csrb_wrapper(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows, const Index columns, const Index used_elements, const int BlockHeight, const int BlockWidth)
{
  //CUSPARSE
  if (BlockHeight == BlockWidth && typeid(unsigned int) == typeid(IT_))
  {
    csrb_intern<DT_, IT_>(r, a, x, b, y, val, col_ind, row_ptr, rows, columns, used_elements, BlockHeight);
  }
  //GENERIC
  else
  {
    csrb_intern<DT_, IT_>(r, a, x, b, y, val, col_ind, row_ptr, rows, columns, used_elements, BlockHeight, BlockWidth);
  }
}
template void Apply<Mem::CUDA>::csrb_wrapper<float, unsigned int>(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index, const int, const int);
template void Apply<Mem::CUDA>::csrb_wrapper<double, unsigned int>(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index, const int, const int);
template void Apply<Mem::CUDA>::csrb_wrapper<float, unsigned long>(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index, const int, const int);
template void Apply<Mem::CUDA>::csrb_wrapper<double, unsigned long>(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index, const int, const int);

template <typename DT_, typename IT_, int BlockSize_>
void Apply<Mem::CUDA>::csrsb(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index rows,
    const Index columns, const Index used_elements)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  if (Math::abs(b) < Math::eps<DT_>())
  {
    MemoryPool<Mem::CUDA>::set_memory(r, DT_(0), /*(transposed?columns:rows)*/ rows * BlockSize_);
  }
  else if (r != y)
  {
    MemoryPool<Mem::CUDA>::copy(r, y, /*(transposed?columns:rows)*/ rows * BlockSize_);
  }

  FEAT::LAFEM::Intern::cuda_apply_csrsb<DT_, IT_, BlockSize_><<<grid, block>>>(r, a, x, b, val, col_ind, row_ptr, rows);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Apply<Mem::CUDA>::csrsb<float, unsigned long, 1>
  (float *, const float, const float *, const float, const float *, const float * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<double, unsigned long, 1>
  (double *, const double, const double *, const double, const double *, const double * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<float, unsigned int, 1>
  (float *, const float, const float *, const float, const float *, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<double, unsigned int, 1>
  (double *, const double, const double *, const double, const double *, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<float, unsigned long, 2>
  (float *, const float, const float *, const float, const float *, const float * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<double, unsigned long, 2>
  (double *, const double, const double *, const double, const double *, const double * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<float, unsigned int, 2>
  (float *, const float, const float *, const float, const float *, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<double, unsigned int, 2>
  (double *, const double, const double *, const double, const double *, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<float, unsigned long, 3>
  (float *, const float, const float *, const float, const float *, const float * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<double, unsigned long, 3>
  (double *, const double, const double *, const double, const double *, const double * const, const unsigned long * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<float, unsigned int, 3>
  (float *, const float, const float *, const float, const float *, const float * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::csrsb<double, unsigned int, 3>
  (double *, const double, const double *, const double, const double *, const double * const, const unsigned int * const, const unsigned int * const, const Index, const Index, const Index);

template <typename DT_, typename IT_>
void Apply<Mem::CUDA>::ell(DT_ * r, const DT_ a, const DT_ * const x, const DT_ b, const DT_ * const y, const DT_ * const val, const IT_ * const col_ind, const IT_ * const cs, const IT_ * const cl, const Index C, const Index rows)
{
  Index blocksize = MemoryPool<Mem::CUDA>::blocksize_spmv;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  if (Math::abs(b) < Math::eps<DT_>())
  {
    MemoryPool<Mem::CUDA>::set_memory(r, DT_(0), rows);
  }
  else if (r != y)
  {
    MemoryPool<Mem::CUDA>::copy(r, y, rows);
  }

  FEAT::LAFEM::Intern::cuda_apply_mv_ell<<<grid, block>>>(r, a, x, b, val, col_ind, cs, cl, rows, C);
#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Apply<Mem::CUDA>::ell(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const Index, const Index);
template void Apply<Mem::CUDA>::ell(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned int * const, const unsigned int * const, const unsigned int * const, const Index, const Index);
template void Apply<Mem::CUDA>::ell(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const Index, const Index);
template void Apply<Mem::CUDA>::ell(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned long * const, const unsigned long * const, const unsigned long * const, const Index, const Index);

template <typename DT_, typename IT_>
void Apply<Mem::CUDA>::banded(DT_ * r, const DT_ alpha, const DT_ * const x, const DT_ beta, const DT_ * const y, const DT_ * const val, const IT_ * const offsets, const Index num_of_offsets, const Index rows, const Index columns)
{
  Index blocksize(128);
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((rows)/(double)(block.x));

  if (Math::abs(beta) < Math::eps<DT_>())
  {
    MemoryPool<Mem::CUDA>::set_memory(r, DT_(0), rows);
  }
  else if (r != y)
  {
    MemoryPool<Mem::CUDA>::copy(r, y, rows);
  }

  FEAT::LAFEM::Intern::cuda_apply_banded<<<grid, block>>>(r, alpha, x, beta, val, offsets, num_of_offsets, rows, columns);
}
template void Apply<Mem::CUDA>::banded(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::banded(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned int * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::banded(float *, const float, const float * const, const float, const float * const, const float * const, const unsigned long * const, const Index, const Index, const Index);
template void Apply<Mem::CUDA>::banded(double *, const double, const double * const, const double, const double * const, const double * const, const unsigned long * const, const Index, const Index, const Index);

template <typename DT_>
void Apply<Mem::CUDA>::dense(DT_ * r, const DT_ alpha, const DT_ beta, const DT_ * const y, const DT_ * const val, const DT_ * const x, const Index rows, const Index columns)
{
  if (r != y)
  {
    hipMemcpy(r, y, rows * sizeof(DT_), hipMemcpyDeviceToDevice);
  }

  FEAT::LAFEM::Intern::cublas_apply_dense(HIPBLAS_OP_T, (int)rows, (int)columns, &alpha, val, x, &beta, r);

#ifdef FEAT_DEBUG_MODE
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
template void Apply<Mem::CUDA>::dense(float * r, const float, const float, const float * const, const float * const, const float * const, const Index, const Index);
template void Apply<Mem::CUDA>::dense(double * r, const double, const double, const double * const, const double * const, const double * const, const Index, const Index);
