// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2021 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/string.hpp>
#include <kernel/util/exception.hpp>


void FEAT::Util::cuda_set_device(const int device)
{
  hipSetDevice(device);
}

void * FEAT::Util::cuda_malloc_host(const Index bytes)
{
  void * memory(nullptr);
  if (bytes == 0)
    return memory;

  if (hipErrorOutOfMemory == hipHostMalloc((void**)&memory, bytes, hipHostMallocMapped))
    throw InternalError(__func__, __FILE__, __LINE__, "MemoryPool<CUDA> cuda pinned allocation error (hipErrorOutOfMemory)");
  if (memory == nullptr)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_malloc_host allocation error (null pointer returned)");
  return memory;
}

void FEAT::Util::cuda_free_host(void * address)
{
  if (address == nullptr)
    return;

  if (hipSuccess != hipHostFree(address))
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_free_host: hipHostFree failed!");
}

void FEAT::Util::cuda_check_last_error()
{
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
}

void * FEAT::Util::cuda_get_device_pointer(void * host)
{
  void * device(nullptr);
  if (hipSuccess != hipHostGetDevicePointer((void**)&device, host, 0))
    throw InternalError(__func__, __FILE__, __LINE__, "hipHostGetDevicePointer failed!");
  return device;
}
