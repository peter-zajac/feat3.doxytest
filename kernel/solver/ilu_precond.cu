#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2023 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>

#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipsparse.h"

// http://docs.nvidia.com/cuda/cusparse/#cusparse-lt-t-gt-csrilu02_solve

using namespace FEAT;

namespace FEAT
{
  namespace Solver
  {
    /// \cond internal
    namespace Intern
    {
      // CSR
      struct CudaIluSolveInfo
      {
        hipsparseMatDescr_t descr_M;
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
        csrilu02Info_t info_M;
        csrsv2Info_t  info_L;
        csrsv2Info_t  info_U;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer;
        int structural_zero;
        int numerical_zero;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        double * z;
        int m;
        int nnz;
      };

      void * cuda_ilu_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd)
      {
        CudaIluSolveInfo * info = new CudaIluSolveInfo;
        info->m = m;
        info->nnz = nnz;

        hipMalloc((void**)&(info->z), m * sizeof(double));


        hipsparseStatus_t status;

        hipsparseCreateMatDescr(&(info->descr_M));
        hipsparseSetMatIndexBase(info->descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(info->descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

        hipsparseCreateMatDescr(&(info->descr_L));
        hipsparseSetMatIndexBase(info->descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(info->descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(info->descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(info->descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&(info->descr_U));
        hipsparseSetMatIndexBase(info->descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(info->descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(info->descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(info->descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        hipsparseCreateCsrilu02Info(&(info->info_M));
        hipsparseCreateCsrsv2Info(&(info->info_L));
        hipsparseCreateCsrsv2Info(&(info->info_U));

        status = hipsparseDcsrilu02_bufferSize(Util::Intern::cusparse_handle, m, nnz,
                info->descr_M, csrVal, csrRowPtr, csrColInd, info->info_M, &(info->pBufferSize_M));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrilu02_bufferSize failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, info->trans_L, m, nnz,
            info->descr_L, csrVal, csrRowPtr, csrColInd, info->info_L, &(info->pBufferSize_L));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseDcsrsv2_bufferSize failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, info->trans_U, m, nnz,
            info->descr_U, csrVal, csrRowPtr, csrColInd, info->info_L, &(info->pBufferSize_U));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseDcsrsv2_bufferSize failed with status code: " + stringify(status));

        info->pBufferSize = max(info->pBufferSize_M, max(info->pBufferSize_L, info->pBufferSize_U));
        hipMalloc((void**)&(info->pBuffer), info->pBufferSize_M);

        status = hipsparseDcsrilu02_analysis(Util::Intern::cusparse_handle, m, nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->info_M,
                    info->policy_M, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrilu02_analysis failed with status code: " + stringify(status));
        status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &(info->structural_zero));
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }

        status = hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, info->trans_L, m, nnz, info->descr_L,
                csrVal, csrRowPtr, csrColInd, info->info_L, info->policy_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparse_csrv_analysis failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, info->trans_U, m, nnz, info->descr_U,
                csrVal, csrRowPtr, csrColInd, info->info_U, info->policy_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparse_csrv_analysis failed with status code: " + stringify(status));

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return (void*)info;
      }

      void cuda_ilu_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipsparseStatus_t status = hipsparseDcsrilu02(Util::Intern::cusparse_handle, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->info_M, info->policy_M, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrilu02 failed with status code: " + stringify(status));
        status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &(info->numerical_zero));
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilu_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;
        const double alpha = 1.;

        hipsparseStatus_t status = hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "sucparseDcsrsv2_solve failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsr2_solve failed with status code: " + stringify(status));

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilu_done_symbolic(void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipFree(info->z);
        hipFree(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
        hipsparseDestroyCsrilu02Info(info->info_M);
        hipsparseDestroyCsrsv2Info(info->info_L);
        hipsparseDestroyCsrsv2Info(info->info_U);

        delete info;
      }

      // BCSR
      struct CudaIluBSolveInfo
      {
        hipsparseMatDescr_t descr_M;
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
        bsrilu02Info_t info_M;
        bsrsv2Info_t info_L;
        bsrsv2Info_t info_U;
        hipsparseOperation_t trans_L;
        hipsparseOperation_t trans_U;
        hipsparseDirection_t dir;
        hipsparseSolvePolicy_t policy_M;
        hipsparseSolvePolicy_t policy_L;
        hipsparseSolvePolicy_t policy_U;
        void * pBuffer;
        double * z;
        int m;
        int nnz;
        int blocksize;
      };

      void * cuda_ilub_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd, const int blocksize)
      {
        double * z;
        hipMalloc((void**)&z, m * blocksize * sizeof(double));

        hipsparseMatDescr_t descr_M = 0;
        hipsparseMatDescr_t descr_L = 0;
        hipsparseMatDescr_t descr_U = 0;
        bsrilu02Info_t info_M  = 0;
        bsrsv2Info_t  info_L  = 0;
        bsrsv2Info_t  info_U  = 0;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer = 0;
        int structural_zero;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;

        hipsparseStatus_t status;

        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        hipsparseCreateBsrilu02Info(&info_M);
        hipsparseCreateBsrsv2Info(&info_L);
        hipsparseCreateBsrsv2Info(&info_U);

        hipsparseDbsrilu02_bufferSize(Util::Intern::cusparse_handle, dir, m, nnz,
                descr_M, csrVal, csrRowPtr, csrColInd, blocksize, info_M, &pBufferSize_M);
        hipsparseDbsrsv2_bufferSize(Util::Intern::cusparse_handle, dir, trans_L, m, nnz,
                descr_L, csrVal, csrRowPtr, csrColInd, blocksize, info_L, &pBufferSize_L);
        hipsparseDbsrsv2_bufferSize(Util::Intern::cusparse_handle, dir, trans_U, m, nnz,
                descr_U, csrVal, csrRowPtr, csrColInd, blocksize, info_U, &pBufferSize_U);

        pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

        hipMalloc((void**)&pBuffer, pBufferSize);

        status = hipsparseDbsrilu02_analysis(Util::Intern::cusparse_handle, dir, m, nnz, descr_M,
                csrVal, csrRowPtr, csrColInd, blocksize, info_M,
                    policy_M, pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrilu02 failed with status code: " + stringify(status));
        status = hipsparseXbsrilu02_zeroPivot(Util::Intern::cusparse_handle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }

        status = hipsparseDbsrsv2_analysis(Util::Intern::cusparse_handle, dir, trans_L, m, nnz, descr_L,
                csrVal, csrRowPtr, csrColInd, blocksize, info_L, policy_L, pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrv2_analysis failed with status code: " + stringify(status));

        status = hipsparseDbsrsv2_analysis(Util::Intern::cusparse_handle, dir, trans_U, m, nnz, descr_U,
                csrVal, csrRowPtr, csrColInd, blocksize, info_U, policy_U, pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrv2_analysis failed with status code: " + stringify(status));

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        CudaIluBSolveInfo * info = new CudaIluBSolveInfo;
        info->descr_M = descr_M;
        info->descr_L = descr_L;
        info->descr_U = descr_U;
        info->info_M  = info_M;
        info->info_L  = info_L;
        info->info_U  = info_U;
        info->trans_L = trans_L;
        info->trans_U = trans_U;
        info->dir = dir;
        info->policy_M = policy_M;
        info->policy_L = policy_L;
        info->policy_U = policy_U;
        info->pBuffer = pBuffer;
        info->z = z;
        info->m = m;
        info->nnz = nnz;
        info->blocksize = blocksize;

        return (void*)info;
      }

      void cuda_ilub_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;

        hipsparseStatus_t status = hipsparseDbsrilu02(Util::Intern::cusparse_handle, info->dir, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_M, info->policy_M, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrilu02 failed with status code: " + stringify(status));
        int numerical_zero;
        status = hipsparseXbsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &numerical_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilub_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;
        const double alpha = 1.;

        hipsparseStatus_t status = hipsparseDbsrsv2_solve(Util::Intern::cusparse_handle, info->dir, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrsv2_solve failed with status code: " + stringify(status));

        status = hipsparseDbsrsv2_solve(Util::Intern::cusparse_handle, info->dir, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrsv2_solve failed with status code: " + stringify(status));

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilub_done_symbolic(void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;

        hipFree(info->z);
        hipFree(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
        hipsparseDestroyBsrilu02Info(info->info_M);
        hipsparseDestroyBsrsv2Info(info->info_L);
        hipsparseDestroyBsrsv2Info(info->info_U);

        delete info;
      }
    } // namespace Intern
    /// \endcond
  } // namespace Solver
} // namespace FEAT
